
#include <hip/hip_runtime.h>
/***
#include "SolverGPU.cuh"
#include "Common.cuh"
#include "Common.hpp"
#include "Core/Timestep.h"

__device__ __constant__ SimParams deviceSimParams;
SimParams hostSimParams;

__device__ inline void AtomicAdd(glm::vec3 *address, int index, glm::vec3 val, int reorder) {
    const int r1 = reorder % 3;
    const int r2 = (reorder + 1) % 3;
    const int r3 = (reorder + 2) % 3;
    atomicAdd(&(address[index].x) + r1, val[r1]);
    atomicAdd(&(address[index].x) + r2, val[r2]);
    atomicAdd(&(address[index].x) + r3, val[r3]);
}

void SetSimParams(SimParams* hostParams)
{
    ScopedTimerGPU timer("Solver_SetParams");
    checkCudaErrors(cudaMemcpyToSymbolAsync(d_params, hostParams, sizeof(VtSimParams)));
    h_params = *hostParams;
}
***/